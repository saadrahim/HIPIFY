// RUN: %run_test hipify "%s" "%t" %hipify_args -D__CUDA_API_VERSION_INTERNAL %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main() {
  printf("04. CUDA Driver API Defines synthetic test\n");

  // CHECK: #define __HIPCC__;
  #define __HIPCC__;

  // CHECK: int DEVICE_CPU = hipCpuDeviceId;
  // CHECK-NEXT: int DEVICE_INVALID = hipInvalidDeviceId;
  // CHECK-NEXT: int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;
  int DEVICE_CPU = hipCpuDeviceId;
  int DEVICE_INVALID = hipInvalidDeviceId;
  int IPC_HANDLE_SIZE = HIP_IPC_HANDLE_SIZE;

  // CHECK: void* LAUNCH_PARAM_BUFFER_POINTER = HIP_LAUNCH_PARAM_BUFFER_POINTER;
  // CHECK-NEXT: void* LAUNCH_PARAM_BUFFER_SIZE = HIP_LAUNCH_PARAM_BUFFER_SIZE;
  // CHECK-NEXT: void* LAUNCH_PARAM_END = HIP_LAUNCH_PARAM_END;
  void* LAUNCH_PARAM_BUFFER_POINTER = HIP_LAUNCH_PARAM_BUFFER_POINTER;
  void* LAUNCH_PARAM_BUFFER_SIZE = HIP_LAUNCH_PARAM_BUFFER_SIZE;
  void* LAUNCH_PARAM_END = HIP_LAUNCH_PARAM_END;

  // CHECK: int MEMHOSTALLOC_PORTABLE = hipHostMallocPortable;
  // CHECK-NEXT: int MEMHOSTALLOC_DEVICEMAP = hipHostMallocMapped;
  // CHECK-NEXT: int MEMHOSTALLOC_WRITECOMBINED = hipHostMallocWriteCombined;
  // CHECK-NEXT: int MEMHOSTREGISTER_PORTABLE = hipHostRegisterPortable;
  // CHECK-NEXT: int MEMHOSTREGISTER_DEVICEMAP = hipHostRegisterMapped;
  // CHECK-NEXT: int MEMHOSTREGISTER_IOMEMORY = hipHostRegisterIoMemory;
  int MEMHOSTALLOC_PORTABLE = hipHostMallocPortable;
  int MEMHOSTALLOC_DEVICEMAP = hipHostMallocMapped;
  int MEMHOSTALLOC_WRITECOMBINED = hipHostMallocWriteCombined;
  int MEMHOSTREGISTER_PORTABLE = hipHostRegisterPortable;
  int MEMHOSTREGISTER_DEVICEMAP = hipHostRegisterMapped;
  int MEMHOSTREGISTER_IOMEMORY = hipHostRegisterIoMemory;

  // CHECK: int TRSA_OVERRIDE_FORMAT = HIP_TRSA_OVERRIDE_FORMAT;
  // CHECK-NEXT: int TRSF_NORMALIZED_COORDINATES = HIP_TRSF_NORMALIZED_COORDINATES;
  // CHECK-NEXT: int TRSF_READ_AS_INTEGER = HIP_TRSF_READ_AS_INTEGER;
  // CHECK-NEXT: int TRSF_SRGB = HIP_TRSF_SRGB;
  int TRSA_OVERRIDE_FORMAT = HIP_TRSA_OVERRIDE_FORMAT;
  int TRSF_NORMALIZED_COORDINATES = HIP_TRSF_NORMALIZED_COORDINATES;
  int TRSF_READ_AS_INTEGER = HIP_TRSF_READ_AS_INTEGER;
  int TRSF_SRGB = HIP_TRSF_SRGB;

  // CHECK: int ARRAY3D_LAYERED = hipArrayLayered;
  // CHECK-NEXT: int ARRAY3D_SURFACE_LDST = hipArraySurfaceLoadStore;
  // CHECK-NEXT: int ARRAY3D_CUBEMAP = hipArrayCubemap;
  // CHECK-NEXT: int ARRAY3D_TEXTURE_GATHER = hipArrayTextureGather;
  int ARRAY3D_LAYERED = hipArrayLayered;
  int ARRAY3D_SURFACE_LDST = hipArraySurfaceLoadStore;
  int ARRAY3D_CUBEMAP = hipArrayCubemap;
  int ARRAY3D_TEXTURE_GATHER = hipArrayTextureGather;

  // CHECK: int COOPERATIVE_LAUNCH_MULTI_DEVICE_NO_PRE_LAUNCH_SYNC = hipCooperativeLaunchMultiDeviceNoPreSync;
  // CHECK-NEXT: int COOPERATIVE_LAUNCH_MULTI_DEVICE_NO_POST_LAUNCH_SYNC = hipCooperativeLaunchMultiDeviceNoPostSync;
  int COOPERATIVE_LAUNCH_MULTI_DEVICE_NO_PRE_LAUNCH_SYNC = hipCooperativeLaunchMultiDeviceNoPreSync;
  int COOPERATIVE_LAUNCH_MULTI_DEVICE_NO_POST_LAUNCH_SYNC = hipCooperativeLaunchMultiDeviceNoPostSync;

  return 0;
}
